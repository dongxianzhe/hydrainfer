#include "hip/hip_runtime.h"
#include<iostream>
#include<torch/torch.h>
#include"reduce_sum.cuh"
#include"dispatch.h"

template<typename T, bool is_even_n>
__global__ void sum_kernel(T* out, T* in, int n){
    if constexpr(std::is_same<T, half>::value && is_even_n){
        half2 val = make_half2(0.f, 0.f);
        const int block_size = blockDim.x;
        for(int i = threadIdx.x;i < n / 2;i += block_size){
            val += reinterpret_cast<half2*>(in)[i];
        }
        val = block_reduce_sum<half2>(val);
        if(threadIdx.x == 0) out[threadIdx.x] = val.x + val.y;
    }
    else{
        T val = 0.f;
        const int block_size = blockDim.x;
        for(int i = threadIdx.x;i < n;i += block_size){
            val += in[i];
        }
        val = block_reduce_sum<T>(val);
        if(threadIdx.x == 0) out[threadIdx.x] = val;
    }
}

void sum(at::Tensor& out, const at::Tensor& in){
    int n_elements = in.numel();
    TORCH_CHECK(in.scalar_type() == out.scalar_type());
    TORCH_CHECK(out.numel() == 1);
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE(in.scalar_type(), scalar_t, [&]{
        DISPATCH_BOOL(n_elements % 2 == 0, is_even_n, [&]{
            dim3 gridDim(1);
            dim3 blockDim;
            if constexpr(std::is_same<scalar_t, half>::value && is_even_n){
                blockDim = min(n_elements / 2, 1024);
            }else{
                blockDim = min(n_elements, 1024);
            }
            sum_kernel<scalar_t, is_even_n><<<gridDim, blockDim>>>(
                static_cast<scalar_t*>(out.data_ptr()), 
                static_cast<scalar_t*>(in.data_ptr()), 
                n_elements
            );
        });
    });
}