#include "hip/hip_runtime.h"
#include<iostream>
#include<gtest/gtest.h>
#include<torch/torch.h>
#include<cute/layout.hpp>
#include<cute/stride.hpp>
#include<cute/tensor.hpp>

TEST(cute, tensor){
    torch::Tensor t = torch::randn({2, 3});
    std::cout << t << std::endl;

    cute::Tensor tensor = cute::make_tensor(static_cast<float*>(t.data_ptr()), cute::make_shape(cute::Int<2>{}, cute::Int<3>{}), cute::make_stride(cute::Int<3>{}, cute::Int<1>{}));
    cute::print(tensor);std::cout << std::endl;
    
    for (int i = 0; i < cute::size<0>(tensor); ++i) {
        for (int j = 0; j < cute::size<1>(tensor); ++j) {
            tensor(i, j) ++;
        }
    }

    std::cout << t << std::endl;
}

TEST(cute, local_tile){
    using namespace cute;
    torch::Tensor a = torch::randn({1024, 256});
    Tensor t = make_tensor(
        static_cast<float*>(a.data_ptr()), 
        make_shape(Int<1024>{}, Int<256>{}), 
        make_stride(Int<256>{}, Int<1>{})
    );
    print(t);puts("");
    Tensor s1 = local_tile(t, make_tile(Int<128>{}, Int<32>{}), make_coord(_, _));
    Tensor s2 = local_tile(t, make_tile(Int<128>{}, Int<32>{}), make_coord(2, _));
    Tensor s3 = local_tile(t, make_tile(Int<128>{}, Int<32>{}), make_coord(_, 3));
    Tensor s4 = local_tile(t, make_tile(Int<128>{}, Int<32>{}), make_coord(2, 3));
    print(s1);puts("");
    print(s2);puts("");
    print(s3);puts("");
    print(s4);puts("");
}

TEST(cute, slice){
    using namespace cute;
    torch::Tensor a = torch::randn({1024, 256});
    Tensor t = make_tensor(
        static_cast<float*>(a.data_ptr()), 
        make_shape(Int<1024>{}, Int<256>{}), 
        make_stride(Int<256>{}, Int<1>{})
    );
    print(t);puts("");
    Tensor s = local_tile(t, make_tile(Int<128>{}, Int<32>{}), make_coord(2, _));
    print(s);puts("");
    auto s_slice = s(make_coord(_, _, 1));
    print(s_slice);puts("");
}

__global__ void shared_memory_kernel(){
    constexpr int len = 48 * 1024 / 4;
    __shared__ volatile float data[len];
    data[len - 1] = 1;
}

TEST(kernel, max_shared_memory){
    shared_memory_kernel<<<1024, 32>>>();
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
}

template <class Engine1, class Engine2, class Layout>
__device__ void tile_g2s_copy(cute::Tensor<Engine1, Layout> ga, cute::Tensor<Engine2, Layout> sa){
    // a (128, 32) : (x, 1)
    // b (128, 32) : (x, 1)
    // 128 threads
    using namespace cute;
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, half>;

    auto g2s_tiled_copy_a = make_tiled_copy(
        g2s_copy_atom{}, 
        make_layout(
            make_shape(Int<32>{}, Int<4>{}),
            make_stride(Int<4>{}, Int<1>{})
        ), // thread layout
        make_layout(make_shape(Int<1>{}, Int<8>{})) // data tile layout each thread copy
    );

    int idx = threadIdx.x;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto taga_copy = g2s_thr_copy_a.partition_S(ga); // (8, 128 / 32, 32 / 4)
    auto tasa_copy = g2s_thr_copy_a.partition_D(sa); // (8, 128 / 32, 32 / 4)

    cute::copy(g2s_tiled_copy_a, taga_copy(_, _, _), tasa_copy(_, _, _));
    cp_async_fence();
    cp_async_wait<0>();
    __syncthreads();
}

__global__ void g2s_copy_kernel(half* a, half* o){
    using namespace cute;    
    Tensor ga = make_tensor(make_gmem_ptr(a), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor go = make_tensor(make_gmem_ptr(o), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    __shared__ half ashm[128 * 32];
    Tensor sa = make_tensor(make_smem_ptr(ashm), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));

    tile_g2s_copy(ga, sa);

    // s2g
    int i = threadIdx.x;
    for(int j = 0;j < 32;j ++){
        go(i, j) = sa(i, j);
    }
}

TEST(cute, g2s_copy){
    torch::TensorOptions options = torch::dtype(torch::kHalf).device(torch::kCUDA);
    torch::Tensor a = torch::randn({128, 32}, options);
    torch::Tensor o_ref = a.clone();
    torch::Tensor o = torch::zeros({128, 32}, options);
    g2s_copy_kernel<<<1, 128>>>(
        static_cast<half*>(a.data_ptr()), 
        static_cast<half*>(o.data_ptr())
    );
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    std::cout << o_ref.view({-1}).slice(0, 0, 4) << std::endl;
    std::cout << "============================================================" << std::endl;
    std::cout << o.view({-1}).slice(0, 0, 4) << std::endl;
    EXPECT_TRUE(torch::allclose(o, o_ref));
}

TEST(cute, layout){
    using namespace cute;
    auto l = make_layout(make_shape(Int<1>{}, Int<8>{}));
    print(l);
}