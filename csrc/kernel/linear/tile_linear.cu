#include "hip/hip_runtime.h"
#include "tile_linear.h"
#include<torch/torch.h>
#include<cute/layout.hpp>
#include<cute/stride.hpp>
#include<cute/tensor.hpp>

namespace mllm{

__global__ void tile_linear_kernel(half* aptr, half* bptr, half* cptr){
    using namespace cute;
    Tensor ga = make_tensor(make_gmem_ptr(aptr), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor gb = make_tensor(make_gmem_ptr(bptr), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor gc = make_tensor(make_gmem_ptr(cptr), make_shape(Int<128>{}, Int<128>{}), make_stride(Int<128>{}, Int<1>{}));
    __shared__ half ashm [128 * 32];
    __shared__ half bshm [128 * 32];
    __shared__ half cshm [128 * 128];
    Tensor sa = make_tensor(make_smem_ptr(ashm), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor sb = make_tensor(make_smem_ptr(bshm), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor sc = make_tensor(make_smem_ptr(cshm), make_shape(Int<128>{}, Int<128>{}), make_stride(Int<128>{}, Int<1>{}));

    {
    // 1. g2s
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, half>;
    auto thr_layout = make_layout(make_shape(Int<32>{}, Int<4>{}), make_stride(Int<4>{}, Int<1>{}));
    auto val_layout = make_layout(make_shape(Int<1>{}, Int<8>{}), make_stride(Int<8>{}, Int<1>{}));
    auto g2s_tiled_copy = make_tiled_copy(g2s_copy_atom{}, thr_layout, val_layout);
    auto g2s_thr_copy = g2s_tiled_copy.get_slice(threadIdx.x);
    auto g2s_ga = g2s_thr_copy.partition_S(ga); // (8, 128 / 32 / 1, 32 / 4 / 8)
    auto g2s_sa = g2s_thr_copy.partition_D(sa); // (8, 128 / 32 / 1, 32 / 4 / 8)
    auto g2s_gb = g2s_thr_copy.partition_S(gb); // (8, 128 / 32 / 1, 32 / 4 / 8)
    auto g2s_sb = g2s_thr_copy.partition_D(sb); // (8, 128 / 32 / 1, 32 / 4 / 8)
    cute::copy(g2s_tiled_copy, g2s_ga(_, _, _), g2s_sa(_, _, _));
    cute::copy(g2s_tiled_copy, g2s_gb(_, _, _), g2s_sb(_, _, _));
    cp_async_fence();
    cp_async_wait<0>();
    __syncthreads();
    }

    {
    // 2. s2r
    Tensor ra = make_tensor<half>(make_shape(Int<8>{}, Int<2>{}, Int<2>{}), make_stride(Int<1>{}, Int<8>{}, Int<16>{}));
    Tensor rb = make_tensor<half>(make_shape(Int<4>{}, Int<16>{}, Int<2>{}), make_stride(Int<1>{}, Int<4>{}, Int<64>{}));
    Tensor rc = make_tensor<half>(make_shape(Int<4>{}, Int<2>{}, Int<16>{}), make_stride(Int<1>{}, Int<4>{}, Int<8>{}));
    Tensor s2r_sa = make_tensor(
        make_smem_ptr(ashm), 
        make_shape (Int<2>{}, Int<2>{}     , Int<2>{}, Int<2>{}      , Int<2>{} , Int<4>{}, Int<8>{} , Int<4>{}     ), 
        make_stride(Int<1>{}, Int<8 * 32>{}, Int<8>{}, Int<64 * 32>{}, Int<16>{}, Int<2>{}, Int<32>{}, Int<16 * 32>{})
        );
    Tensor s2r_ra = make_tensor(
        make_rmem_ptr(ra.data()),
        make_shape(Int<2>{}, Int<2>{}, Int<2>{}, Int<2>{}, Int<2>{}), 
        make_stride(Int<1>{}, Int<2>{}, Int<4>{}, Int<8>{}, Int<16>{})
        );
    Tensor s2r_sb = make_tensor(
        make_smem_ptr(bshm), 
        make_shape (Int<2>{}, Int<2>{}, Int<16>{}    , Int<2>{} , Int<4>{}, Int<8>{} , Int<4>{}), 
        make_stride(Int<1>{}, Int<8>{}, Int<8 * 32>{}, Int<16>{}, Int<2>{}, Int<32>{}, Int<0>{})
    );
    Tensor s2r_rb = make_tensor(
        make_rmem_ptr(rb.data()), 
        make_shape (Int<2>{}, Int<2>{}, Int<16>{}, Int<2>{}), 
        make_stride(Int<1>{}, Int<2>{}, Int<4>{} , Int<64>{})
    );
    const int warp_id = threadIdx.x / 32;
    const int lane_id = threadIdx.x % 32;
    for(int x = 0;x < 2;x ++){
        for(int y = 0;y < 2;y ++){
            for(int i = 0;i < 2;i ++){
                for(int j = 0;j < 2;j ++){
                    for(int k = 0;k < 2;k ++){
                        s2r_ra(k, j, i, x, y) = s2r_sa(k, j, i, x, y, lane_id % 4, lane_id / 4, warp_id);                   
                    }
                }
            }
        }
    }
    for(int x = 0;x < 16;x ++){
        for(int y = 0;y < 2;y ++){
            for(int i = 0;i < 2;i ++){
                for(int j = 0;j < 2;j ++){
                    s2r_rb(j, i, x, y) = s2r_sb(j, i, x, y, lane_id % 4, lane_id / 4, warp_id);
                }
            }
        }
    }
    clear(rc);
    __syncthreads();
    // 3. compute
    auto mma_atom = SM80_16x8x16_F16F16F16F16_TN{};
    auto thr_layout = make_layout(make_shape(Int<4>{}, Int<1>{}, Int<1>{}));  // m n k
    auto permutations = Tile<Int<64>, Int<32>, Int<16>>{}; // todo is there make_tile?
    auto tiled_mma = make_tiled_mma(mma_atom, thr_layout, permutations);
    for(int ik = 0;ik < 2; ik ++){
        cute::gemm(tiled_mma, rc, ra(_, _, ik), rb(_, _, ik), rc);
    }
    __syncthreads();
    // 4. r2s
    Tensor r2s_rc = make_tensor(
        make_rmem_ptr(rc.data()), 
        make_shape (Int<2>{}, Int<2>{}, Int<2>{}, Int<16>{}), 
        make_stride(Int<1>{}, Int<2>{}, Int<4>{}, Int<8>{} )
        );
    Tensor r2s_sc = make_tensor(
        make_smem_ptr(cshm), 
        make_shape (Int<2>{}, Int<2>{}      , Int<2>{}       , Int<16>{}, Int<4>{}, Int<8>{}  , Int<4>{}       ), 
        make_stride(Int<1>{}, Int<8 * 128>{}, Int<64 * 128>{}, Int<8>{} , Int<2>{}, Int<128>{}, Int<16 * 128>{})
    );
    for(int m = 0;m < 2;m ++){
        for(int n = 0;n < 16;n ++){
            for(int i = 0;i < 2;i ++){
                for(int j = 0;j < 2;j ++){
                    r2s_sc(j, i, m, n, lane_id % 4, lane_id / 4, warp_id) = r2s_rc(j, i, m, n);
                }
            }
        }
    }
    __syncthreads();
    }

    {
    // 5. s2g
    auto s2g_copy_atom = Copy_Atom<UniversalCopy<cute::uint128_t>, half>{};
    auto thr_layout = make_layout(make_shape(Int<4>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    auto val_layout = make_layout(make_shape(Int<1>{}, Int<8>{}), make_stride(Int<8>{}, Int<1>{}));
    auto s2g_tiled_copy = make_tiled_copy(s2g_copy_atom, thr_layout, val_layout);
    auto s2g_thr_copy = s2g_tiled_copy.get_thread_slice(threadIdx.x);
    auto s2g_sc = s2g_thr_copy.partition_S(sc); // (8, 128 / 4 / 1, 128 / 32 / 8)
    auto s2g_gc = s2g_thr_copy.partition_D(gc); // (8, 128 / 4 / 1, 128 / 32 / 8)
    cute::copy(s2g_tiled_copy, s2g_sc(_, _, _), s2g_gc(_, _, _));
    }
}

torch::Tensor tile_linear(torch::Tensor a, torch::Tensor b){
    constexpr int M = 128;
    constexpr int N = 128;
    constexpr int K = 32;

    TORCH_CHECK(a.dtype() == torch::kHalf);
    TORCH_CHECK(b.dtype() == torch::kHalf);
    TORCH_CHECK(a.is_cuda());
    TORCH_CHECK(b.is_cuda());
    TORCH_CHECK(a.is_contiguous());
    TORCH_CHECK(b.is_contiguous());
    TORCH_CHECK(a.dim() == 2);
    TORCH_CHECK(b.dim() == 2);
    TORCH_CHECK(a.size(0) == M);
    TORCH_CHECK(a.size(1) == K);
    TORCH_CHECK(b.size(0) == N);
    TORCH_CHECK(b.size(1) == K);

    torch::TensorOptions options = torch::dtype(torch::kHalf).device(torch::kCUDA);
    auto c = torch::zeros({M, N}, options);

    dim3 gridDim{1};
    dim3 blockDim{128};
    tile_linear_kernel<<<gridDim, blockDim>>>(
        static_cast<half*>(a.data_ptr()), 
        static_cast<half*>(b.data_ptr()),
        static_cast<half*>(c.data_ptr())
    );
    return c;
}

}