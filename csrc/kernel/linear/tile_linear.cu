#include "hip/hip_runtime.h"
#include "tile_linear.h"
#include<torch/torch.h>
#include<cute/layout.hpp>
#include<cute/stride.hpp>
#include<cute/tensor.hpp>

namespace mllm{

__global__ void tile_linear_kernel(half* aptr, half* bptr, half* cptr){
    using namespace cute;
    Tensor ga = make_tensor(make_gmem_ptr(aptr), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor gb = make_tensor(make_gmem_ptr(bptr), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor gc = make_tensor(make_gmem_ptr(cptr), make_shape(Int<128>{}, Int<128>{}), make_stride(Int<128>{}, Int<1>{}));
    __shared__ half ashm [128 * 32];
    __shared__ half bshm [128 * 32];
    __shared__ half cshm [128 * 128];
    Tensor sa = make_tensor(make_smem_ptr(ashm), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor sb = make_tensor(make_smem_ptr(bshm), make_shape(Int<128>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
    Tensor sc = make_tensor(make_smem_ptr(cshm), make_shape(Int<128>{}, Int<128>{}), make_stride(Int<128>{}, Int<1>{}));
    // 1. g2s
    {
        using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
        using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
        using g2s_copy_atom = Copy_Atom<g2s_copy_traits, half>;
        auto thr_layout = make_layout(make_shape(Int<32>{}, Int<4>{}), make_stride(Int<4>{}, Int<1>{}));
        auto val_layout = make_layout(make_shape(Int<1>{}, Int<8>{}), make_stride(Int<8>{}, Int<1>{}));
        auto g2s_tiled_copy = make_tiled_copy(g2s_copy_atom{}, thr_layout, val_layout);
        auto g2s_thr_copy = g2s_tiled_copy.get_slice(threadIdx.x);
        auto g2s_ga = g2s_thr_copy.partition_S(ga); // (8, 128 / 32 / 1, 32 / 4 / 8)
        auto g2s_sa = g2s_thr_copy.partition_D(sa); // (8, 128 / 32 / 1, 32 / 4 / 8)
        auto g2s_gb = g2s_thr_copy.partition_S(gb); // (8, 128 / 32 / 1, 32 / 4 / 8)
        auto g2s_sb = g2s_thr_copy.partition_D(sb); // (8, 128 / 32 / 1, 32 / 4 / 8)
        cute::copy(g2s_tiled_copy, g2s_ga(_, _, _), g2s_sa(_, _, _));
        cute::copy(g2s_tiled_copy, g2s_gb(_, _, _), g2s_sb(_, _, _));
        cp_async_fence();
        cp_async_wait<0>();
        __syncthreads();
    }
    // 2. compute
    {
        int i = threadIdx.x;
        for(int j = 0;j < 128;j ++){
            half sum = 0;
            for(int k = 0;k < 32;k ++){
                sum += ga(i, k) * gb(j, k);
            }
            sc(i, j) = sum;
        }
        __syncthreads();
    }
    // 3. s2g
    {
        auto s2g_copy_atom = Copy_Atom<UniversalCopy<cute::uint128_t>, half>{};
        auto thr_layout = make_layout(make_shape(Int<4>{}, Int<32>{}), make_stride(Int<32>{}, Int<1>{}));
        auto val_layout = make_layout(make_shape(Int<1>{}, Int<8>{}), make_stride(Int<8>{}, Int<1>{}));
        auto s2g_tiled_copy = make_tiled_copy(s2g_copy_atom, thr_layout, val_layout);
        auto s2g_thr_copy = s2g_tiled_copy.get_thread_slice(threadIdx.x);
        auto s2g_sc = s2g_thr_copy.partition_S(sc); // (8, 128 / 4 / 1, 128 / 32 / 8)
        auto s2g_gc = s2g_thr_copy.partition_D(gc); // (8, 128 / 4 / 1, 128 / 32 / 8)
        cute::copy(s2g_tiled_copy, s2g_sc(_, _, _), s2g_gc(_, _, _));
    }
}

torch::Tensor tile_linear(torch::Tensor a, torch::Tensor b){
    constexpr int M = 128;
    constexpr int N = 128;
    constexpr int K = 32;

    TORCH_CHECK(a.dtype() == torch::kHalf);
    TORCH_CHECK(b.dtype() == torch::kHalf);
    TORCH_CHECK(a.is_cuda());
    TORCH_CHECK(b.is_cuda());
    TORCH_CHECK(a.is_contiguous());
    TORCH_CHECK(b.is_contiguous());
    TORCH_CHECK(a.dim() == 2);
    TORCH_CHECK(b.dim() == 2);
    TORCH_CHECK(a.size(0) == M);
    TORCH_CHECK(a.size(1) == K);
    TORCH_CHECK(b.size(0) == N);
    TORCH_CHECK(b.size(1) == K);

    torch::TensorOptions options = torch::dtype(torch::kHalf).device(torch::kCUDA);
    auto c = torch::zeros({M, N}, options);

    dim3 gridDim{1};
    dim3 blockDim{128};
    tile_linear_kernel<<<gridDim, blockDim>>>(
        static_cast<half*>(a.data_ptr()), 
        static_cast<half*>(b.data_ptr()),
        static_cast<half*>(c.data_ptr())
    );
    return c;
}

}